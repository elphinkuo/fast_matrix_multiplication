// Filename: matrixMul.cu

#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 1024
#define BLOCK_SIZE 16

__global__ void matrixMulCUDA(float *C, float *A, float *B) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0;
    if (col < N && row < N) {
        for (int i = 0; i < N; i++) {
            sum += A[row * N + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

// Host function to execute the matrix multiplication
void matrixMul(float *A, float *B, float *C) {
    float *d_A, *d_B, *d_C;

    // Allocate device memory
    hipMalloc((void **)&d_A, N * N * sizeof(float));
    hipMalloc((void **)&d_B, N * N * sizeof(float));
    hipMalloc((void **)&d_C, N * N * sizeof(float));

    // Copy matrices from the host to device
    hipMemcpy(d_A, A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Setup the execution configuration
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid((N + dimBlock.x - 1) / dimBlock.x, (N + dimBlock.y - 1) / dimBlock.y);

    // Launch the CUDA Kernel
    matrixMulCUDA<<<dimGrid, dimBlock>>>(d_C, d_A, d_B);

    // Copy the result matrix back to the host
    hipMemcpy(C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
